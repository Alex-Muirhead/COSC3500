#include "hip/hip_runtime.h"
// COSC3500, Semester 2, 2021
// Assignment 2
// Main file - AVX version

#include "eigensolver.h"
#include "randutil.h"
#include <string>
#include <iostream>
#include <iomanip>
#include <immintrin.h>

#define ALIGN 64
#define _mm256_from_ptr(ptr) *((__m256d*)(ptr))

// global variables to store the matrix

double* M = nullptr;
double* Mdevice = nullptr;

double* Ydevice = nullptr;
double* Xdevice = nullptr;

int N = 0;
int Blocks = 0;

const int BLOCKSIZE = 256;

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

// 59 us @ N=100, 558 us @ N=1000, 5658 us @ N=10_000
// __global__
// void MatVecMulKernel(int N, double* Y, const double* X, const double* M)
// {
//    // Each thread calculates a single value of Y.
//    int row = blockIdx.x*BLOCKSIZE + threadIdx.x;
//    double Yvalue = 0.0;
//    __shared__ double Xsub[BLOCKSIZE];
//    for (int subIdx = 0; subIdx < gridDim.x; ++subIdx)
//    {
//       int precol = threadIdx.x + subIdx*BLOCKSIZE;
//       if (precol < N)
//          Xsub[threadIdx.x] = X[precol];
//       __syncthreads();
//       for (int subcol = 0; subcol < BLOCKSIZE; ++subcol)
//       {
//          int col = subcol + subIdx*BLOCKSIZE;
//          if (col >= N || row >= N) break;
//          Yvalue += M[col*N+row] * Xsub[subcol];
//       }
//       __syncthreads();
//    }
//    if (row < N)
//       Y[row] = Yvalue;
// }

// 50 us @ N=100, 254 us @ N=1000, 4036 us @ N=10_000
__global__
void MatVecMulKernel(int N, double* Y, const double* X, const double* M)
{
   // Each thread calculates a single value of Y.
   int i = blockIdx.x*BLOCKSIZE + threadIdx.x;
   if (i >= N) return;
   double Yvalue = 0.0;
   for (int j = 0; j < N; ++j)
   {
      Yvalue += M[j*N+i] * X[j];
   }
   Y[i] = Yvalue;
}

// implementation of the matrix-vector multiply function
void MatrixVectorMultiply(double* Y, const double* X)
{
   // Total threads = Blocks * ThreadsPerBlock >= N
   // At least one thread per row of matrix, avoids data-race at Y
   checkError(hipMemcpy(Xdevice, X, N*sizeof(double), hipMemcpyHostToDevice));
   MatVecMulKernel<<<Blocks, BLOCKSIZE>>>(N, Ydevice, Xdevice, Mdevice);
   checkError(hipMemcpy(Y, Ydevice, N*sizeof(double), hipMemcpyDeviceToHost));
}

int main(int argc, char** argv)
{
   // get the current time, for benchmarking
   auto StartTime = std::chrono::high_resolution_clock::now();

   // get the input size from the command line
   if (argc < 2)
   {
      std::cerr << "expected: matrix size <N>\n";
      return 1;
   }
   N = std::stoi(argv[1]);

   Blocks = (N+BLOCKSIZE-1)/BLOCKSIZE;

   // Allocate memory for the matrix
   M = static_cast<double*>(malloc(N*N*sizeof(double)));
   checkError(hipMalloc(&Mdevice, N*N*sizeof(double)));

   // Allocate CUDA memory space for vectors
   checkError(hipMalloc(&Ydevice, N*sizeof(double)));
   checkError(hipMalloc(&Xdevice, N*sizeof(double)));

   // seed the random number generator to a known state
   randutil::seed(4);  // The standard random number.  https://xkcd.com/221/

   // Initialize the matrix.  This is a matrix from a Gaussian Orthogonal Ensemble.
   // The matrix is symmetric.
   // The diagonal entries are gaussian distributed with variance 2.
   // The off-diagonal entries are gaussian distributed with variance 1.
   for (int i = 0; i < N; ++i)
   {
      M[i*N+i] = std::sqrt(2.0) * randutil::randn();
      for (int j = i+1; j < N; ++j)
      {
         M[i*N + j] = M[j*N + i] = randutil::randn();
      }
   }
   checkError(hipMemcpy(Mdevice, M, N*N*sizeof(double), hipMemcpyHostToDevice));
   auto FinishInitialization = std::chrono::high_resolution_clock::now();

   // Call the eigensolver
   EigensolverInfo Info = eigenvalues_arpack(N, 100);

   auto FinishTime = std::chrono::high_resolution_clock::now();

   auto InitializationTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishInitialization - StartTime);
   auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishTime - StartTime);

   std::cout << "Obtained " << Info.Eigenvalues.size() << " eigenvalues.\n";
   std::cout << "The largest eigenvalue is: " << std::setw(16) << std::setprecision(12) << Info.Eigenvalues.back() << '\n';
   std::cout << "Total time:                             " << std::setw(12) << TotalTime.count() << " us\n";
   std::cout << "Time spent in initialization:           " << std::setw(12) << InitializationTime.count() << " us\n";
   std::cout << "Time spent in eigensolver:              " << std::setw(12) << Info.TimeInEigensolver.count() << " us\n";
   std::cout << "   Of which the multiply function used: " << std::setw(12) << Info.TimeInMultiply.count() << " us\n";
   std::cout << "   And the eigensolver library used:    " << std::setw(12) << (Info.TimeInEigensolver - Info.TimeInMultiply).count() << " us\n";
   std::cout << "Total serial (initialization + solver): " << std::setw(12) << (TotalTime - Info.TimeInMultiply).count() << " us\n";
   std::cout << "Number of matrix-vector multiplies:     " << std::setw(12) << Info.NumMultiplies << '\n';
   std::cout << "Time per matrix-vector multiplication:  " << std::setw(12) << (Info.TimeInMultiply / Info.NumMultiplies).count() << " us\n";

   // free memory
   checkError(hipFree(Xdevice));
   checkError(hipFree(Ydevice));
   checkError(hipFree(Mdevice));
   free(M);
}
