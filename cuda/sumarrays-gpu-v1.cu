
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

// code that will run on the GPU, but can call it from the CPU
__global__
void add(int n, double* x, double const* y)
{
   int index = threadIdx.x;
   int stride = blockDim.x;     // blockDim is the number of threads in a block
   for (int i = index; i < n; i += stride)
   {
      x[i] = x[i] + y[i];
   }
}

int main()
{
   int N = 1<<20; // pow(2,20) = 1,048,576

   // allocate memory on the host
   double* x = new double[N];
   double* y = new double[N];

   // initialize arrays
   for (int i = 0; i < N; i++)
   {
      x[i] = 1.0;
      y[i] = 2.0;
   }

   // allocate memory on the device
   double* xDevice;
   double* yDevice;
   checkError(hipMalloc(&xDevice, N*sizeof(double)));
   checkError(hipMalloc(&yDevice, N*sizeof(double)));

   // copy memory from host to device
   checkError(hipMemcpy(xDevice, x, N*sizeof(double), hipMemcpyHostToDevice));
   checkError(hipMemcpy(yDevice, y, N*sizeof(double), hipMemcpyHostToDevice));

   auto t1 = std::chrono::high_resolution_clock::now();
   // Invoke the CUDA kernel with add<<<NumberOfBlocks, NumberOfThreadsPerBlock>>>
   add<<<1, 1>>>(N, xDevice, yDevice); // asynchronous
   checkError(hipDeviceSynchronize());
   auto t2 = std::chrono::high_resolution_clock::now();

   // copy memory from device back to host
   checkError(hipMemcpy(x, xDevice, N*sizeof(double), hipMemcpyDeviceToHost));

   // check the result
   for (int i = 0; i < N; ++i)
   {
      if (x[i] != 3.0)
      {
         std::cerr << "error at array index " << i << " value " << x[i] << " expected 3.0\n";
         std::abort();
      }
   }

   auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2-t1).count();
   std::cout << "Time = " << duration << " us\n";

   // clean up
   checkError(hipFree(xDevice));
   checkError(hipFree(yDevice));
   delete[] x;
   delete[] y;
}
